#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define spaceSize 1000
#define numBoids 1024 //has to be multiple of blockSize^2
#define blockSize 32
#define numIters 1000
#define visualRange 200
#define boidMass 1
#define maxSpeed 100
#define minDistance 5
#define centerAttrWeight 0.05
#define repulsionWeight 0.5
#define alignmentWeight 0.05

void checkCudaError(int id)
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error(id=%d): %s\n", id, hipGetErrorString(error));
    }
}

__host__ __device__ struct Boid
{
    double x;
    double y;
    double xVel;
    double yVel;
    double xAcc;
    double yAcc;
}

__host__ __device__ struct Force
{
    double Fx;
    double Fy;
}

// Helper function to calculate distance between two boids
__device__ double calcDistance(Boid& b1, Boid& b1)
{
    return sqrt((b1.x - b2.x) * (b1.x - b2.x) + (b1.y - b2.y) * (b1.y - b2.y));
}

// Helper function to apply a force to a boid
__device__ void applyForce(Boid& boid, Force& force)
{
    boid.xAcc += force.Fx / boidMass;
    boid.yAcc += force.Fy / boidMass;
}

// Apply the attraction force to center and applies it to boid
__device__ void applyCenterAttr(Boid& boid, int currIdx, Boid* boids, int nBoids)
{
    // Find the center of mass
    double xSum = 0;
    double ySum = 0;
    int count = 0;
    for(int i = 0; i < nBoids; i++)
    {
        if(calcDistance(boids[currIdx], boids[i]) <= visualRange && i != currIdx)
        {
            xSum += boids[i].x;
            ySum += boids[i].y;
            count++;
        }
    }
    if(count == 0)
    {
        return;
    }
    // Calculate the force
    xSum /= count;
    ySum /= count;
    double distance = sqrt((xSum - b1.x) * (xSum - b1.x) + (ySum - b1.y) * (ySum - b1.y));
    double sinTheta = (ySum - b1.y) / distance;
    double cosTheta = (xSum - b1.x) / distance;
    Force force = {centerAttrWeight * count * cosTheta / distance, 
                    centerAttrWeight * count * sinTheta / distance};
    applyForce(boids[currIdx], force);
}

// Apply the repulsion force to avoid other boids
__device__ void applyAvoidOthers(Boid& boid, int currIdx, Boid* boids, int nBoids)
{
    for(int i = 0; i < nBoids; i++)
    {
        if(calcDistance(boids[currIdx], boids[i]) < minDistance && i != currIdx)
        {
            double distance = calcDistance(boids[currIdx], boids[i]);
            double sinTheta = (boids[i].y - boids[currIdx].y) / distance;
            double cosTheta = (boids[i].x - boids[currIdx].x) / distance;
            Force force = {repulsionWeight * cosTheta * (distance - minDistance), 
                            repulsionWeight * sinTheta * (distance - minDistance)};
            applyForce(boids[currIdx], force);
        }
    }
}

// Apply the alignment force to make this velocity match the central velocity
__device__ void applyAlignment(Boid& boid, int currIdx, Boid* boids, int nBoids)
{
    double vXSum = 0;
    double vYSum = 0;
    int count = 0;
    for(int i = 0; i < nBoids; i++)
    {
        if(calcDistance(boids[currIdx], boids[i]) <= visualRange && i != currIdx)
        {
            vXSym += boids[i].xVel;
            vYSum += boids[i].yVel;
            count++;
        }
    }
    if(count == 0)
    {
        return;
    }
    // Align the velocity slightly to average velocity
    vXSum /= count;
    vYSum /= count;
    Force force = {(vXSum - boids[currIdx].xVel)*alignmentWeight, 
                    (vYSum - boids[currIdx].yVel)*alignmentWeight};
    applyForce(boids[currIdx], force);
}

// Helper function at the end of each iteration to update position based off velocity
// LIMITS SPEED
// BOUNCES OFF WALLS
// RESETS ACCELERATION
__device__ void updateBoid(Boid& boid)
{
    // UPDATE POSITIONS
    boid.x += boid.xVel;
    boid.y += boid.yVel;
    // If hit wall, bounce off
    if(boid.x < 0 || boid.x > spaceSize)
    {
        boid.xVel *= -1;
        if(boid.x < 0)
        {
            boid.x = -1 * boid.x;
        }
        else
        {
            boid.x = spaceSize - (boid.x - spaceSize);
        }
    }
    if(boid.y < 0 || boid.y > spaceSize)
    {
        boid.yVel *= -1;
        if(boid.y < 0)
        {
            boid.y = -1 * boid.y;
        }
        else
        {
            boid.y = spaceSize - (boid.y - spaceSize);
        }
    }
    // UPDATE SPEED
    boid.xVel += boid.xAcc;
    boid.yVel += boid.yAcc;
    // Limit speed
    double speed = sqrt(boid.xVel * boid.xVel + boid.yVel * boid.yVel);
    if(speed > maxSpeed)
    {
        boid.xVel = boid.xVel * maxSpeed / speed;
        boid.yVel = boid.yVel * maxSpeed / speed;
    }
    // UPDATE ACCELERATION
    boid.xAcc = 0;
    boid.yAcc = 0;
}

// One step of naive: calculate the acceleration of each boid. DOESN'T apply it yet
__global__ void naiveCalcAcc(boid* boids, int nBoids)
{
    int currIdx = blockIdx.x * blockDim.x + threadIdx.x;
    Boid boidCpy = boids[currIdx];
    // Center attarction force
    // NOTE: could probably optimize by making memory accesses closer to each other
    applyCenterAttr(boidCpy, currIdx, boids, nBoids);
    applyAvoidOthers(boidCpy, currIdx, boids, nBoids);
    applyAlignment(boidCpy, currIdx, boids, nBoids);
    boids[currIdx] = boidCpy;
}

__global__ void naiveUpdateBoids(Boid* boids, int nBoids)
{
    int currIdx = blockIdx.x * blockDim.x + threadIdx.x;
    Boid boidCpy = boids[currIdx];
    updatePosition(boidCpy);
    boids[currIdx] = boidCpy;
}

int main(int argc, char **argv)
{
    // Allocate memory for host
    Boid *boids;
    // Paged-locked memory doesn't get swapped back to disk
    hipHostMalloc((void**)&boids, sizeof(Boid) * numBoids);
	
    // Initialize boids
    int gap = spaceSize / numBoids;
    for(int i = 0; i < numBoids; i++)
    {
        boids[i].x = i * gap;
        boids[i].y = i * gap;
        boids[i].xVel = random() % 20 - 10;
        boids[i].yVel = random() % 20 - 10;
        boids[i].xAcc = 0;
        boids[i].yAcc = 0;
    }

    // Allocate memory for device
    Boid* gpu_boids;
    hipMalloc(&gpu_boids, sizeof(Boid) * numBoids);
    
    // create events and streams
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    std::ofstream ofile("output.txt");

    // Start calling the gpu
    hipEventRecord(startEvent, 0);
    dim3 dimBlock(blockSize);
    dim3 dimGrid(numBoids/blockSize);
    // Run all the timesteps
    hipMemcpy(gpu_boids, boids, sizeof(Boid) * numBoids, hipMemcpyHostToDevice);
    for(int i = 0; i < numIters; i++)
    {
        naiveCalcAcc <<< dimGrid, dimBlock >>> (gpu_boids, numBoids);
        checkCudaError(0);
        naiveUpdateBoids <<< dimGrid, dimBlock >>> (gpu_boids, numBoids);
        hipMemcpy(boids, gpu_boids, sizeof(Boid) * numBoids, hipMemcpyDeviceToHost);
        checkCudaError(0);
        // Print out the all the boids
        ofile << "ITERATION " << i << "\n";
        for(int j = 0; j < numBoids; j++)
        {
            ofile << "Boid " << j << ": " << boids[j].x << ", " << boids[j].y << "\n";
        }
    }
    ofile.close();

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float ms;
    hipEventElapsedTime(&ms, startEvent, stopEvent);

    printf("time is %f ms\n", ms);
    printf("boids[50].x is %f: \n", final_boids[50].x);

    // Free the memory
    hipHostFree(boids);
    hipFree(gpu_boids);
    return 0;
}